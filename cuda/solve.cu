#include "hip/hip_runtime.h"
/*
  Copyright 2009-2016 Lianqi Wang <lianqiw-at-tmt-dot-org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "solve.h"
namespace cuda_recon{
Real cucg_t::solve(curcell &xout, const curcell &xin, stream_t &stream){
    Real ans;
    cgtmp.count++;
    if((ans=gpu_pcg(xout, this, precond, xin, cgtmp,
		    warm_restart, maxit, stream))>1){
	cgtmp.count_fail++;
	warning2("CG %5d(%5d) does not converge. residual=%g. maxit=%d\n", 
		 cgtmp.count, cgtmp.count_fail, ans, maxit);
    }
    return ans;
}

void cumuv_t::Forward(curcell &out, Real beta, const curcell &in, Real alpha, stream_t &stream){
    if(!M) error("M Can not be empty\n");
    if(!out){
	out=curcell(nx, 1, nxs, (int*)NULL);
    }else{
	curscale(out.M(), beta, stream);
    }
    cuspmul(out.M().P(), M, in.M().P(), 1, 'n', alpha, stream);
    if(U && V){
	curmv(Vx.P(), 0, V, in.M().P(), 't', 1, stream);
	curmv(out.M().P(), 1, U, Vx.P(), 'n', -alpha, stream);
    }
}
void cumuv_t::Trans(curcell &out, Real beta, const curcell &in, Real alpha, stream_t &stream){
    if(!M) error("M Can not be empty\n");
    if(!out){
	out=curcell(ny, 1, nys, (int*)NULL);
    }else{
	curscale(out.M(), beta, stream);
    }
    
    curscale(out.M(), beta, stream);
    cuspmul(out.M().P(), M, in.M().P(), 1, 't', alpha, stream);
    if(U && V){
	curmv(Vx.P(), 0, U, in.M().P(), 't', 1, stream);
	curmv(out.M().P(), 1, V, Vx.P(), 'n', -alpha, stream);
    }
}
void cumuv_t::Init(const MUV_T *in){
    if(!in) return;
    if(M || !in->M) error("in.M() should not be NULL and M should be NULL\n");
    dspcell *inM=dspcell_cast(in->M);
    dsp *Mc=dspcell2sp(inM);
    dmat *Uc=dcell2m(in->U);
    dmat *Vc=dcell2m(in->V);
    nx=inM->nx;
    ny=inM->ny;
    nxs=new int[nx];
    nys=new int[ny];
    for(int i=0; i<nx; i++){
	nxs[i]=inM->p[i]->nx;
    }
    for(int i=0; i<ny; i++){
	nys[i]=inM->p[i*inM->nx]->ny;
    }
    M=cusp(Mc, 1);
    cp2gpu(U, Uc);
    cp2gpu(V, Vc);
    dspfree(Mc); dfree(Uc); dfree(Vc);
    Vx=curmat(V.Ny(), 1);
}

cusolve_sparse::cusolve_sparse(int _maxit, int _warm_restart, MUV_T *_R, MUV_T *_L)
    :cucg_t(_maxit, _warm_restart){
    CR.Init(_R);
    CL.Init(_L);
}
cusolve_cbs::cusolve_cbs(spchol *_C, dmat *_Up, dmat *_Vp){
    if(!_C){
	error("C cannot be empty\n");
    }
    chol_convert(_C, 0);
    Cl=cusp(_C->Cl, 0);
    cp2gpu(Cp, _C->Cp, _C->Cl->nx, 1);
    if(_Up){
	cp2gpu(Up, _Up);
	cp2gpu(Vp, _Vp);
    }
}
Real cusolve_cbs::solve(curcell &xout, const curcell &xin, stream_t &stream){
    if(!xout) xout=xin.New();
    if(Cl.Type()==SP_CSC){
	chol_solve(xout.M().P(), xin.M().P(), stream);
    }else{
	error("To implemente\n");
    }
    if(Up){
	if(!Vr){
	    Vr=curmat(Vp.Ny(), 1);
	}
	curmv(Vr.P(), 0, Vp, xin.M().P(), 't', -1, stream);
	curmv(xout.M().P(), 1, Up, Vr.P(), 'n', 1, stream);
    }
    return 0;
}
/*solve in place*/
static __global__ void cuchol_solve_lower_do(Real *restrict y, Real *Cx, int *Cp, int *Ci, int n){
    int id=threadIdx.x;
    int nd=blockDim.x;
    extern __shared__ Real sb[];
    __shared__ Real val;
    /*first solve L\y*/
    
    for(int icol=0; icol<n; icol++){
	if(id==0){
	    y[icol]/=Cx[Cp[icol]];//divide by diagonal.
	    val=-y[icol];
	}
	__syncthreads();//this is necessary!
	for(int irow=Cp[icol]+1+id; irow<Cp[icol+1]; irow+=nd){
	    y[Ci[irow]]+=val*Cx[irow];
	}
	__syncthreads();
    }
    /*Next solve L'\y. Use reduction algorithm instead of atomic add.*/
    for(int icol=n-1; icol>-1; icol--){
	sb[id]=0;
	for(int irow=Cp[icol]+1+id; irow<Cp[icol+1]; irow+=nd){
	    sb[id]+=Cx[irow]*y[Ci[irow]];
	}
	for(int step=(blockDim.x>>1);step>0;step>>=1){
	    __syncthreads();
	    if(id<step){
		sb[id]+=sb[id+step];
	    }
	}
	if(id==0){
	    y[icol]=(y[icol]-sb[0])/Cx[Cp[icol]];
	}
	__syncthreads();//this is necessary!
    }
}
void cusolve_cbs::chol_solve(Real *out, const Real *in, stream_t &stream){
    if(!Cl || !Cp) error("Invalid\n");
    int n=Cl.Nx();
    if(!y){
	y=curmat(Cl.Nx(), 1);
    }
    perm_f_do<<<DIM(n, 256),0,stream>>>(y.P(), in, Cp.P(), n);
    //only 1 block for synchronization. //todo: improve the implementation.
    const int NTH=256;
    cuchol_solve_lower_do<<<1,NTH, NTH*sizeof(Real),stream>>>(y.P(), Cl.Px(), Cl.Pp(), Cl.Pi(), n); 
    perm_i_do<<<DIM(n, 256),0,stream>>>(out, y.P(), Cp.P(), n);
    hipStreamSynchronize(stream);
}
}
