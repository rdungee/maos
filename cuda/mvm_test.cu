#include "hip/hip_runtime.h"
/*
  Copyright 2009-2016 Lianqi Wang <lianqiw-at-tmt-dot-org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "utils.h"
#include "curmat.h"
#include "cucmat.h"

/**
\file mvm_test.cu

Test performance of different cuda implementations for matrix vector multiplication.
Make sure the results is correct first.
Test using transpose.

*/
typedef Real ATYPE;
typedef Real GTYPE;

#define tix threadIdx.x
#define tiy threadIdx.y
#define bix blockIdx.x
#define biy blockIdx.y
#define tnx blockDim.x
#define tny blockDim.y
#define bnx gridDim.x
#define bny gridDim.y

#define BLOCKMV_TNY 1
#define BLOCKMV_TNX 256
/*
  y=A*x;
  Call with
  blockmv<<<ngrid, dim(tnx, tny), (tny+tnx*tny)*sizeof(Real), stream>>>()
  with both tnx and tny must be power of two and be less than 32. tny must be 32.
*/

__global__ static void
blockmv(Real *restrict y, const Real *restrict A, const Real *restrict x, const int nrow, const int ncol){
    __shared__ Real shared[BLOCKMV_TNX][BLOCKMV_TNY];
    //Real *sx=shared;//stores x in shared memory
    Real *sy=&shared[tix][tiy];//stores non-reduced y in shared memory, transposed for easy reduction.
    const int irow=(tnx*bix+tix);
    *sy=0;
#if 1
    if(irow<nrow){
	for(int icol=tiy; icol<ncol; icol+=tny){//a few pass
	    *sy+=A[irow+icol*nrow]*x[icol];
	}
    }
    //printf("0 (%02d %02d) =%g\n", tix, tiy, *sy);
#else
    for(int ip=0; ip<ncol; ip+=tny){//a few pass
	const int icol=ip+tiy;
	if(tix==0){
	    if(icol<ncol){
		sx[tiy]=x[icol];
	    }else{
		sx[tiy]=0;
	    }
	}
	__syncthreads();
	if(irow<nrow){
	    *sy+=A[irow+icol*nrow]*sx[tiy];
	}
    }
#endif
    /*
#if BLOCKMV_TNY==32
    if(irow<nrow){
#pragma unroll
	for(int step=(BLOCKMV_TNY>>1); step>0; step>>=1){
	    __syncthreads();
	    if(tiy<step){
		sy[0]+=sy[step];
	    }
	}
	if(!tiy){
	    atomicAdd(&y[irow], sy[0]);
	}
    }
#else
    __syncthreads();
    if(tiy==0 && irow<nrow){
#pragma unroll
	for(int i=1; i<BLOCKMV_TNY; i++){
	    sy[0]+=sy[i];
	}
        atomicAdd(&y[irow], sy[0]);
    }
    #endif*/
}
/**
  Another version to increase the occpancy. 
  multimv<<<ngrid, dim(tnx, tny), (tnx*tny)*sizeof(Real), stream>>>()
  tny can be any number. tnx is ~256/tny.
*/
/*__global__ static void
multimv(Real *restrict y, const Real *restrict A, const Real *restrict x, const int nrow, const int ncol){

}*/
__global__ static void
test_read(Real *A, int nx, int ny){
    //extern __shared__ Real sh[];
    const int irow=tnx*bix+tix;
    Real __shared__ sum;
    sum=0;
    for(int i=0; i<ny; i++){
	sum+=A[irow+i*nx];
    }
}
__global__ static void
test_read_multi(Real *A, int nx, int ny){
    //extern __shared__ Real sh[];
    int irow=tnx*bix+tix;
    const int nset=(tnx*bnx+nx-1)/nx;
    const int iset=irow/nx;
    irow=irow-iset*nx;
    Real __shared__ sum;
    sum=0;
    for(int i=iset*ny/nset; i<(iset+1)*ny/nset; i++){
	sum+=A[irow+i*nx];
    }
}

void mvm_test(int igpu){
    hipSetDevice(igpu);
    int M,N;
    M=6981;
    N=2895*2;
    //N=2048*2;
    //N=4000;
    int nstream=5;
    //M=32*32*32;
    //N=600;
    //M=32;
    //N=32*32;
    int iN=1200;
    iN=N;
    X(mat) *mvm=X(new)(M,N);
    X(mat) *x=X(new)(N,1);
    rand_t stat;
    seed_rand(&stat, 1);
    srandn(mvm, 1, &stat);
    srandn(x, 1, &stat);
    //writebin(mvm, "mvm");
    //writebin(x, "x");
    X(mat) *mvmt=strans(mvm);
    curmat *cumvmt=NULL;
    curmat *cumvm=NULL;
    curmat *cux=NULL, *cuy=NULL;
    cp2gpu(&cumvm, mvm);
    cp2gpu(&cumvmt, mvm);
    cp2gpu(&cux, x);
    cuy=curnew(M, 1);
    stream_t stream[nstream];
    int nevent=nstream*2;
    event_t event[nevent];
    Real one=1.;
    //cuwrite(cuy, "y0");
    Real tm;
    hipProfilerStart();
    
    {
	event[0].record(stream[0]);
	for(int i=0; i<N; i+=iN){
	    int nleft=N-i;
	    if(nleft>iN) nleft=iN;
	    DO(CUBL(gemv)(stream[0], HIPBLAS_OP_N, M,nleft, &one, cumvm->p+i*M, M, cux->p+i, 1, &one, cuy->p, 1));
	}
	event[1].record(stream[0]);
	stream[0].sync();
	DO(hipEventElapsedTime(&tm, event[0], event[1]));
	info("cublas?gemv takes %.6f ms\n", tm);
	cuwrite(cuy, "y_cugemv");
    }
    {
	for(int nover=2; nover<256; nover+=2)
	    {
		//custom method. 10 is optimal
		int naeach=128;
		const int nblock=(M*nover+naeach-1)/naeach;
		cuzero(cuy);
		int is;
		for(is=0; is<nstream; is++){
		    stream[is].sync();
		}
		TIC;tic;
		//event[0].record(stream[0]);
		is=0;
		for(int i=0; i<N; i+=iN){
		    int nleft=N-i;
		    if(nleft>iN) nleft=iN;
		    multimv_do<<<nblock, naeach, sizeof(Real)*naeach, stream[is]>>>
			(cumvm->p+i*M, cuy->p, cux->p+i, M, nleft);
		    is=(is+1)%nstream;
		}
		//event[1].record(stream[0]);
		for(is=0; is<nstream; is++){
		    stream[is].sync();
		}
		info("nover=%d,  %.6f ms\n", nover, toc3*1000);
	    }
	cuwrite(cuy, "y_multimv");
    }
    {
	//custom method
	int naeach, mp_count;
	naeach=128;
	mp_count=(M+naeach-1)/naeach;
	cuzero(cuy, stream[0]);stream[0].sync();
    	event[0].record(stream[0]);
	for(int i=0; i<N; i+=iN){
	    int nleft=N-i;
	    if(nleft>iN) nleft=iN;
	    mvm_do<<<mp_count, naeach, sizeof(Real)*naeach, stream[0]>>>
		(cumvm->p+i*M, cuy->p, cux->p+i, M, nleft);
	}
	event[1].record(stream[0]);
	stream[0].sync();
	DO(hipEventElapsedTime(&tm, event[0], event[1]));
	info("mvm_g_mul takes %.6f ms\n", tm);
	cuwrite(cuy, "y_gmul");
	}
    /*  {
	
	const int nblock=(M+BLOCKMV_TNX-1)/BLOCKMV_TNX;
	//const int nblock=1;
	//new method with blocking
	cuzero(cuy, stream[0]);stream[0].sync();
	hipProfilerStart();
	event[0].record(stream[0]);
	for(int i=0; i<N; i+=iN){
	    int nleft=N-i;
	    if(nleft>iN) nleft=iN;
	    blockmv<<<nblock, dim3(BLOCKMV_TNX, BLOCKMV_TNY), 0, stream[0]>>>
		(cuy->p, cumvm->p+i*M, cux->p+i, M, nleft);
	}
	event[1].record(stream[0]);
	stream[0].sync();
	hipProfilerStop();
	DO(hipEventElapsedTime(&tm, event[0], event[1]));
	info("blockmv takes %.6f ms\n", tm);
	cuwrite(cuy, "y_blockmv");
	}*/
    /*{
	hipProfilerStart();
	int nthread=192;
	int nblock=M/nthread;
	//TIC;tic;
	for(int is=0; is<nstream; is++){
	    event[2*is].record(stream[is]);
	    test_read<<<nblock, nthread, 0, stream[is]>>>
		(cumvm->p+M*N/nstream*is, M, N/nstream);
	    event[2*is+1].record(stream[is]);
	}
	for(int is=0; is<nstream; is++){
	    stream[is].sync();
	}
	//toc("sync");
	hipProfilerStop();
	for(int is=0; is<nstream; is++){
	    Real tm1, tm2;
	    DO(hipEventElapsedTime(&tm1, event[0], event[2*is]));
	    DO(hipEventElapsedTime(&tm2, event[0], event[2*is+1]));
	    info2("%.6f %.6f %.6f\n", tm1, tm2, tm2-tm1);
	}
	DO(hipEventElapsedTime(&tm, event[0], event[nevent-1]));
	info("test_read takes %.6f ms, BW=%.1f GB/s\n", tm, M*N*4/tm*1e-6);

	}*/

    /*{
	
	hipProfilerStart();
	const int ncase=100;
	int nthread=128;//192 or 256 are good values
	for(N=4000; N>10; N=N/2){
	    int i0;
	    Real t0=INFINITY;
	    for(nstream=2; nstream<=512; nstream+=2){
		int nblock=M/nthread*nstream;
		Real tm0=0;
		for(int icase=0; icase<ncase; icase++){
		    //A single stream, but different kernels does different columns.
		    //nstream=256;
		    event[0].record(stream[0]);
		    test_read_multi<<<nblock, nthread, 0, stream[0]>>>
			(cumvm->p, M, N);
		    event[1].record(stream[0]);
		    stream[0].sync();
		    DO(hipEventElapsedTime(&tm, event[0], event[1]));
		    tm0+=tm;
		}
		if(tm0<t0){
		    t0=tm0;
		    i0=nstream;
		}
	
	    }
info2("N=%d nstream=%d, BW=%g\n", N, i0, M*N*4/t0*1e-6*ncase);
	}
	hipProfilerStop();
    }*/
}
