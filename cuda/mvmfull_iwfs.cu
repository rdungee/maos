#include "hip/hip_runtime.h"
/*
  Copyright 2009-2016 Lianqi Wang <lianqiw-at-tmt-dot-org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <errno.h>
#include "utils.h"
#include "curmat.h"
#include "cucmat.h"
/**
\file mvmfull_iwfs.cu

Test MVM for a single WFS. Using two GPUs. Optional using networking. ethtest is the server
This is not part of maos executable. Called by test/test_gpu executable.


two ways: 
1) broadcast gradients to both GPUs with each GPU handle part of actuators. (not tested yet)
2) partition gradients to GPUs with GPU handle all actuators. (selected)

Use schedtool -a 0x1 PID to let the exe only run one CPU 0. This prevents jitter.

Optimal timing (low jitter) achieved with multimv_do:
cassiopeia with GTX 580:nsm=1, mtch_ngrid=90, nover=9 1.1ms
orion with single GTX 590: nsm=1, mtch_ngrid=90, nover=9, 1.3ms
kepler with K20: nsm=2, mtch_ngrid=20, nover=3 1.13ms
cassiopeia with single GTX590
*/

#define TIMING 0

#if TIMING 
static unsigned int event_flag=hipEventDefault;
#else
static unsigned int event_flag=hipEventDisableTiming;
#endif
typedef struct{
    curmat cumvm;//active mvm control matrix
    curmat cumvm_next;//inactive mvm control matrix.
    curmat cumvm1;
    curmat cumvm2;
    curmat mtch;
    curmat pix;//pixels. Each sa has 15x6=90 pixels.
    curmat grad;
    curmat act;
    stream_t stream_p;//pixels
    stream_t stream_g;//grads
    cuarray<stream_t> stream_a;//act
    stream_t stream_mvm;//mvm
    int ism;//index of stream for mvm
    int count;
    int gpu;//Which GPU this data is for
    int istep;//Which time step we are in
    int copy_mvm;//1: need to copy mvm.
    int ic;//the column that we are copying.
    hipEvent_t *event_p;
    hipEvent_t *event_g;
    hipEvent_t event_pall;
    event_t *event_w;
#if TIMING
    hipEvent_t event0;
    hipEvent_t *event0_p;
    hipEvent_t *event0_g;
    hipEvent_t event0_a;
    hipEvent_t event_a;
    hipEvent_t *event0_a2;
    hipEvent_t *event_a2;
    hipEvent_t event0_mvm;
    hipEvent_t event_mvm;
#endif
}GPU_DATA_T;
/*Does matched filter*/
static void __global__ mtch_do(const Real *mtch, const Real *pix, Real *grad, int pixpsa, int nsa){
    extern __shared__ Real cum[];//for cumulation and reduction
    Real *cumi=cum+threadIdx.y*blockDim.x;//2 padding for easy reduction
    for(int ig=threadIdx.y+blockDim.y*blockIdx.x; ig<nsa*2; ig+=blockDim.y*gridDim.x){
	const Real *mtchi=mtch+ig*pixpsa;
	const Real *pixi=pix+ig/2*pixpsa;
	//sum 3 times for 90 pixels.
	cumi[threadIdx.x]=0;
	if(threadIdx.x<30){
	    cumi[threadIdx.x]=mtchi[threadIdx.x]*pixi[threadIdx.x]
		+mtchi[threadIdx.x+30]*pixi[threadIdx.x+30]
		+mtchi[threadIdx.x+60]*pixi[threadIdx.x+60];
	}
	//reduction
	for(int step=16;step>0;step>>=1){
	    if(threadIdx.x<step){
		cumi[threadIdx.x]+=cumi[threadIdx.x+step];
	    }
	}
	if(threadIdx.x==0){
	    grad[ig]=cumi[0];
	}
    }
}
/*
__global__ static void 
multimv_do(const Real *restrict mvm, ATYPE *restrict a, const GTYPE *restrict g, int nact, int ng){
    extern __shared__ Real acc[];
    int iact=threadIdx.x+blockIdx.x*blockDim.x;
    int nset=(blockDim.x*gridDim.x+nact-1)/nact;
    if(blockDim.x*gridDim.x<nset*nact){
	//drop partial set
	nset--;
    }
    const int iset=iact/nact;
    if(iset>=nset) return;
    iact=iact-nact*iset;
    acc[threadIdx.x]=0;
    const int igi=(iset*ng)/nset;
    const int ngi=((iset+1)*ng)/nset;
    for(int ig=igi; ig<ngi; ig++){
	Real mvmi=mvm[nact*ig+iact];
	acc[threadIdx.x]+=mvmi*(Real)(g[ig]);
    }
    atomicAdd(&a[iact], (ATYPE)acc[threadIdx.x]);
    }*/

/*__global__ static void mvm_g_mul_do(const Real *restrict mvm, ATYPE *restrict a, const GTYPE *restrict g, int nact, int ng){
    extern __shared__ Real acc[];
    int iact=threadIdx.x+blockIdx.x*blockDim.x;
    if(iact<nact){
	acc[threadIdx.x]=0;
	for(int ig=0; ig<ng; ig++){
	    Real mvmi=mvm[nact*ig+iact];
	    acc[threadIdx.x]+=mvmi*(Real)(g[ig]);
	}
	a[iact]+=(ATYPE)acc[threadIdx.x];
    }
    }*/

/**
   A standalone routine that testes applying MVM for a single WFS and update mvm.
   The orderig of gradients are like xyxyxy instead of normal xxxyyy.

   Important: 
   1) Only page locked host memory can do async memcpy that overallps with computation
   2) Has to be Portable for multiple GPUs to do async memcpy concurrently.
   
*/
void mvmfull_iwfs(int *gpus, int ngpu, int nstep){
    info("Using %d gpus. nstep=%d\n", ngpu, nstep);
    int nstep0=20;//for warm up
#if 1
    //const int nact=7673;//total
    const int nact=6981;//active
    const int nsa=2896;//total. all subaps transported to GPU.
#else
    const int nact=6981;//active
    const int nsa=2700;//active
#endif
    int ng=nsa*2;
    const int pixpsa=90;//Change this need to change kernel mtch_do
    const int pixpsa2=71;//average number of pixels, used for 10GbE
    X(mat) *mvm1=X(new)(nact, ng);
    X(mat) *mvm2=X(new)(nact, ng);
    X(mat) *pix1=X(new)(pixpsa, nsa);
    X(mat) *pix2=X(new)(pixpsa, nsa);
    X(mat) *mtch=X(new)(pixpsa, ng);
    rand_t srand;
    seed_rand(&srand, 1);
    X(randu)(mvm1,1, &srand);
    X(randu)(mvm2,1,&srand);
    X(randu)(pix1,50, &srand);
    X(randu)(pix2,50, &srand);
    X(mat) *mvm=mvm1;
    X(mat) *pix=pix2;
    X(cell) *dmres=X(cellnew)(ngpu, 1);
    X(pagelock)(pix1, pix2, mvm1, mvm2, mtch, dmres, NULL);

    int port=20000;
    int sock=-1;
    int ready=1;

    int mtch_ngrid=30;//50;//30;//can change to utilize GPU fully. 16 is good for cassiopeia
    const int mtch_dimx=32;//must launch 32 threads so that they belong to single wrap.
    const int mtch_dimy=12;//4 subapertures, 8 gradients
    const int naeach=128;
    int nover=14;
    int nsm=2;
    {
	char *MVM_NSM=getenv("MVM_NSM");
	if(MVM_NSM){
	    nsm=strtol(MVM_NSM, NULL, 10);
	    info2("nsm is set to %d\n", nsm);
	}
	char *MVM_NGRID=getenv("MVM_NGRID");
	if(MVM_NGRID){
	    mtch_ngrid=strtol(MVM_NGRID, NULL, 10);
	    info2("mtch_ngrid is set to %d\n", mtch_ngrid);
	}
	char *MVM_NOVER=getenv("MVM_NOVER");
	if(MVM_NOVER){
	    nover=strtol(MVM_NOVER, NULL, 10);
	}
    }
    const int sastep=mtch_dimy*mtch_ngrid/2;
    {
	char *MVM_CLIENT=getenv("MVM_CLIENT");
	if(MVM_CLIENT){
	    char *MVM_PORT=getenv("MVM_PORT");
	    if(MVM_PORT){
		port=strtol(MVM_PORT, NULL, 10);
	    }
	    info2("Connecting to server %s\n", MVM_CLIENT);
	    sock=connect_port(MVM_CLIENT, port, 0 ,1);
	    if(sock!=-1) {
		info2("Connected");
		int cmd[7];
		cmd[0]=nact;
		cmd[1]=nsa;
		cmd[2]=sastep;
		cmd[3]=pixpsa2;
		cmd[4]=nstep;
		cmd[5]=nstep0;
		cmd[6]=1;
		if(stwriteintarr(sock, cmd, 7)){
		    close(sock); sock=-1;
		    warning("Failed: %s\n", strerror(errno));
		}
	    } else {
		info2("Failed\n");
	    }
	}
    }


    int nc=10;//each time copy nc column of mvm.
    GPU_DATA_T **data=new GPU_DATA_T*[ngpu];
    const int sect_gpu=(nsa+sastep*ngpu-1)/(sastep*ngpu);
    for(int igpu=0; igpu<ngpu; igpu++){
	hipSetDevice(gpus[igpu]);
	data[igpu]=new GPU_DATA_T;
	data[igpu]->cumvm1=curmat(mvm1->nx, ng);
	data[igpu]->cumvm2=curmat(mvm2->nx, ng);
	data[igpu]->cumvm=data[igpu]->cumvm1;
	data[igpu]->cumvm_next=data[igpu]->cumvm2;
	cp2gpu(data[igpu]->cumvm1, mvm);
	data[igpu]->pix=curmat(pixpsa, nsa);
	data[igpu]->mtch=curmat(pixpsa, nsa*2);
	cp2gpu(data[igpu]->mtch, mtch);
	data[igpu]->grad=curmat(ng, 1);
	data[igpu]->act=curmat(mvm1->nx, 1);
	data[igpu]->event_w=new event_t[nsm];
	data[igpu]->stream_a=cuarray<stream_t>(nsm);
	data[igpu]->gpu=gpus[igpu];
#if TIMING
	hipEventCreateWithFlags(&data[igpu]->event0, event_flag);
	data[igpu]->event0_g=new hipEvent_t[sect_gpu];
	data[igpu]->event0_p=new hipEvent_t[sect_gpu];
	data[igpu]->event0_a2=new hipEvent_t[sect_gpu];
	data[igpu]->event_a2=new hipEvent_t[sect_gpu];

	for(int i=0; i<sect_gpu; i++){
	    hipEventCreateWithFlags(&data[igpu]->event0_g[i],event_flag);
	    hipEventCreateWithFlags(&data[igpu]->event0_p[i],event_flag);
	    hipEventCreateWithFlags(&data[igpu]->event0_a2[i],event_flag);
	    hipEventCreateWithFlags(&data[igpu]->event_a2[i],event_flag);
	}
	hipEventCreateWithFlags(&data[igpu]->event0_mvm,event_flag);
	hipEventCreateWithFlags(&data[igpu]->event_mvm,event_flag);
	hipEventCreateWithFlags(&data[igpu]->event0_a,event_flag);
	hipEventCreateWithFlags(&data[igpu]->event_a,event_flag);
#endif
	data[igpu]->event_g=new hipEvent_t[sect_gpu];
	data[igpu]->event_p=new hipEvent_t[sect_gpu];
	for(int i=0; i<sect_gpu; i++){
	    hipEventCreateWithFlags(&data[igpu]->event_g[i],event_flag);
	    hipEventCreateWithFlags(&data[igpu]->event_p[i],event_flag);
	}
	hipEventCreateWithFlags(&data[igpu]->event_pall,event_flag);
	dmres->p[igpu]=X(new)(nact, 1);
	X(pagelock)(dmres->p[igpu], NULL);
	/*
	DO(hipMemcpyAsync(data[igpu]->pix->p, pix->p, 2*nsa*pixpsa,
			   hipMemcpyHostToDevice, *data[igpu]->stream_p));
	hipMemcpyAsync(dmres->p[igpu]->p, data[igpu]->act->p, nact*sizeof(Real), 
			hipMemcpyDeviceToHost, data[igpu]->stream_a[0]);
	CUDA_SYNC_DEVICE;
	*/
    }
    X(mat) *timing=X(new)(nstep, 1);
    X(mat) *timing_tot=X(new)(nstep, 1);
    X(mat) *timing_sock=X(new)(nstep, 1);
    X(mat) *result=X(new)(nstep, 1);
    hipProfilerStart();
    TIC;
    if(sock!=-1 && stwriteint(sock, ready)){
	warning("error send ready signal: %s\n", strerror(errno));
	close(sock); sock=-1;
    }
    info2("Ready\n");
    int nblock;
    for(int jstep=-nstep0; jstep<nstep; jstep++){
	//run 20 frames to warm up before timing.
	int istep=jstep<0?0:jstep;
	if(sock!=-1){//start signal
	    timing_sock->p[istep]=0;
	}
	if(nover>0){
	    nblock=(nact*nover+naeach-1)/naeach;
	}else{
	    nblock=(nact*(1+istep/50)+naeach-1)/naeach;
	}
	tic;
#if TIMING
	if(istep%8000==0)
#else
	    if(0)
#endif
	    {//need to update MVM
		if(mvm==mvm1){//switch mvm on host.
		    mvm=mvm2;
		}else{
		    mvm=mvm1;
		}
		for(int igpu=0; igpu<ngpu; igpu++){
		    data[igpu]->copy_mvm=1;
		    if(data[igpu]->ic!=0){
			warning("Sync error, skip update request at step %d\n", istep);
		    }
		}
	    }
	for(int igpu=0; igpu<ngpu; igpu++){
	    data[igpu]->ism=-1;
	    data[igpu]->count=0;
	    data[igpu]->istep=istep;
#if TIMING
	    //beginning of each GPU operation.
	    DO(hipEventRecord(data[igpu]->event0, data[igpu]->stream_a[0]));
#endif
	}
	if(sock==-1){
	    if(pix==pix1){
		pix=pix2;
	    }else{
		pix=pix1;
	    }
	}
	for(int isa=0, igpu=0; isa<nsa; isa+=sastep, igpu=((igpu+1)%ngpu)){
	    hipSetDevice(gpus[igpu]); 
	    GPU_DATA_T *datai=data[igpu];
	    int nleft=(nsa-isa)<sastep?(nsa-isa):sastep;
	    //One stream handling the memcpy
#if TIMING
	    DO(hipEventRecord(datai->event0_p[datai->count], datai->stream_p[0]));
#endif
	    void *pcur=pix->p+isa*pixpsa;
	    if(sock!=-1){
		//pcur=pix->p;//temporary. always use the same buffer
		//manually use 2 byte.
		double tmp0=myclockd();
		if(stread(sock, pcur, 2*nleft*pixpsa2)){
		    warning("failed: %s\n", strerror(errno));
		    close(sock); sock=-1;
		}
		timing_sock->p[istep]+=myclockd()-tmp0;
	    }
	    DO(hipMemcpyAsync(datai->pix.P()+isa*pixpsa, pcur, 2*nleft*pixpsa,
			       hipMemcpyHostToDevice, datai->stream_p));
	    //Recored the event when the memcpy is finished
	    DO(hipEventRecord(datai->event_p[datai->count], datai->stream_p));
	    //Start matched filter when pixel transfer is done.
	    DO(hipStreamWaitEvent(datai->stream_g, datai->event_p[datai->count], 0));
#if TIMING
	    DO(hipEventRecord(datai->event0_g[datai->count], datai->stream_g));    
#endif
	    mtch_do<<<mtch_ngrid, dim3(mtch_dimx, mtch_dimy), 
		mtch_dimx*mtch_dimy*sizeof(Real), datai->stream_g>>>
		(datai->mtch.P()+isa*2*pixpsa, datai->pix.P()+isa*pixpsa, 
		 datai->grad.P()+isa*2, pixpsa, nleft);
	    //Record the event when matched filter is done
	    DO(hipEventRecord(datai->event_g[datai->count], datai->stream_g));

	    //Another stream does the matrix vector multiplication. Wait for the event before executing.
	    //The stream stream will wait only for the completion of the most recent host call to hipEventRecord() on event
	    datai->ism=(datai->ism+1)%nsm;
	    
	    hipStreamWaitEvent(datai->stream_a[datai->ism], datai->event_g[datai->count], 0);
#if TIMING
	    DO(hipEventRecord(datai->event0_a2[datai->count], datai->stream_a[datai->ism]));    
#endif
#if 0
	    DO(CUBL(gemv)(datai->stream_a[datai->ism], HIPBLAS_OP_N, nact, nleft*2, &one, datai->cumvm->p+nact*isa*2, nact, datai->grad->p+isa*2, 1, &one, datai->act->p, 1));
#else
	    multimv_do<<<nblock, naeach, sizeof(Real)*naeach, datai->stream_a[datai->ism]>>>
		(datai->cumvm.P()+nact*isa*2, datai->act.P(), datai->grad.P()+isa*2, 
		 nact, nleft*2);
#endif
	    DO(hipEventRecord(datai->event_w[datai->ism], datai->stream_a[datai->ism]));
#if TIMING
	    DO(hipEventRecord(datai->event_a2[datai->count], datai->stream_a[datai->ism])); 
#endif
	    datai->count++;
	}
	for(int igpu=0; igpu<ngpu; igpu++){
	    GPU_DATA_T *datai=data[igpu];
	    //Record an event when pixel tranporting is over. So we can start transporting mvm matrix.
	    DO(hipEventRecord(datai->event_pall, datai->stream_p));
	}
	//Queue copying MVM matrix to second slot.
	for(int igpu=0; igpu<ngpu; igpu++){
	    GPU_DATA_T *datai=data[igpu];
	    if(datai->copy_mvm){
		int done=0, nleft;
		if(mvm->ny-datai->ic < nc){
		    done=1;
		    nleft=mvm->ny-datai->ic;
		}else{
		    nleft=nc;
		}
		//wait for mvm application to finish before copying.
		//DO(hipStreamWaitEvent(datai->stream_mvm[0], datai->event_pall, 0));
#if TIMING
		DO(hipEventRecord(datai->event0_mvm, datai->stream_mvm[0]));	
#endif
		DO(hipMemcpyAsync(datai->cumvm_next.P()+datai->ic*mvm->nx, 
				   mvm->p+datai->ic*mvm->nx, sizeof(Real)*mvm->nx*nleft, 
				   hipMemcpyHostToDevice, datai->stream_mvm));
#if TIMING
		DO(hipEventRecord(datai->event_mvm, datai->stream_mvm));
#endif
		datai->ic+=nleft;
		if(done){
		    datai->ic=0;
		    datai->copy_mvm=0;
		    curmat tmp=datai->cumvm;
		    datai->cumvm=datai->cumvm_next;
		    datai->cumvm_next=tmp;
		    info2("gpu %d switched over at step %d\n", datai->gpu, datai->istep);
		}
	    }
	}
	//Copy DM commands back to CPU
	for(int igpu=0; igpu<ngpu; igpu++){
	    GPU_DATA_T *datai=data[igpu];
	    hipSetDevice(gpus[igpu]); 
	    for(int ism=1; ism<nsm; ism++){
		DO(hipStreamWaitEvent(datai->stream_a[0], datai->event_w[ism], 0));
	    }
#if TIMING
	    DO(hipEventRecord(datai->event0_a, datai->stream_a[0]));
#endif
	    hipMemcpyAsync(dmres->p[igpu]->p, datai->act.P(), nact*sizeof(Real), hipMemcpyDeviceToHost, datai->stream_a[0]);
#if TIMING
	    DO(hipEventRecord(datai->event_a, datai->stream_a[0]));//record event when all act are copied so mvm can start.
#endif
	}
	//CPU sums them together. sync first gpu
	data[0]->stream_a[0].sync();
	//sum other GPUs
	for(int igpu=1; igpu<ngpu; igpu++){
	    hipSetDevice(gpus[igpu]); 
	    data[igpu]->stream_a[0].sync();
	    for(int iact=0; iact<nact; iact++){
		dmres->p[0]->p[iact]+=dmres->p[igpu]->p[iact];
	    }
	}
	if(sock!=-1){
	    double tmp0=myclockd();
	    if(stwrite(sock, dmres->p[0]->p, sizeof(Real)*nact)){
		warning("error write dmres: %s\n", strerror(errno));
		close(sock); sock=-1;
	    }
	    if(streadint(sock, &ready)){//acknowledgement.
		warning("error read ack failed: %s\n", strerror(errno));
		close(sock), sock=-1;
	    }
	    timing_sock->p[istep]+=myclockd()-tmp0;
	    timing->p[istep]=ready*1.e-6;
	}else{
	    timing->p[istep]=toc3;//do not tic.
	}
	result->p[istep]=dmres->p[0]->p[nact/2];
	usleep(50);//yield
	/*info2("\rStep %d takes %.0f us", istep, timing->p[istep]*1e6);
	if(timing->p[istep]>1.25e-3){
	    info2("\n");
	}	
	*/
	//Wait for MVM matrix copy to finish and time.
	for(int igpu=0; igpu<ngpu; igpu++){
	    GPU_DATA_T *datai=data[igpu];
	    hipSetDevice(datai->gpu);
	    hipMemsetAsync(datai->act.P(), 0, nact*sizeof(Real), datai->stream_a[datai->ism]);
	    datai->stream_a[datai->ism].sync();
	    datai->stream_mvm.sync();
	}
	timing_tot->p[istep]=toc3;
#if TIMING 
	if(istep<100){
	    for(int igpu=0; igpu<ngpu; igpu++){
		hipSetDevice(gpus[igpu]); 
		GPU_DATA_T *datai=data+igpu;
		const int count=datai->count;
		X(mat) *tim=X(new)(count*6+4,2);
		PX(MAT)(tim,ptim);
		int ic;
		for(ic=0; ic<count; ic++){
		    hipEventElapsedTime(&ptim[0][ic*6+0], datai->event0, datai->event0_p[ic]);//start of mtch
		    hipEventElapsedTime(&ptim[0][ic*6+1], datai->event0, datai->event_p[ic]);//end of mtch
		    hipEventElapsedTime(&ptim[0][ic*6+2], datai->event0, datai->event0_g[ic]);//start of g
		    hipEventElapsedTime(&ptim[0][ic*6+3], datai->event0, datai->event_g[ic]);//end of g
		    hipEventElapsedTime(&ptim[0][ic*6+4], datai->event0, datai->event0_a2[ic]);//start of mvm
		    hipEventElapsedTime(&ptim[0][ic*6+5], datai->event0, datai->event_a2[ic]);//end of mvm
		    ptim[1][ic*6]=1;
		    ptim[1][ic*6+1]=1;
		    ptim[1][ic*6+2]=2;
		    ptim[1][ic*6+3]=2;
		    ptim[1][ic*6+4]=3;
		    ptim[1][ic*6+5]=3;
		}
		hipEventElapsedTime(&ptim[0][ic*6+0], datai->event0, datai->event0_a);//start of a copy
		hipEventElapsedTime(&ptim[0][ic*6+1], datai->event0, datai->event_a);//end of a copy
		hipEventElapsedTime(&ptim[0][ic*6+2], datai->event0, datai->event0_mvm);//start of mvm copy
		hipEventElapsedTime(&ptim[0][ic*6+3], datai->event0, datai->event_mvm);//end of mvm copy
		ptim[1][ic*6+0]=4;
		ptim[1][ic*6+1]=4;
		ptim[1][ic*6+2]=5;
		ptim[1][ic*6+3]=5;
		writebin(tim, "timing2_%dgpu%d_step%d", ngpu, igpu, istep);
		X(free)(tim);
	    }
	}
#endif
    }
    hipProfilerStop();
    //writebin(dmres->p[0], "dmres");

    writebin(timing, "timing_%s_%dgpu", HOST, ngpu);
    writebin(timing_tot, "timing_tot_%s_%dgpu", HOST, ngpu);
    writebin(timing_sock, "timing_sock_%s_%dgpu", HOST, ngpu);
    X(pageunlock)(pix1, pix2, mvm1, mvm2, NULL);
    
    X(free)(mvm1);
    X(free)(mvm2);
    X(free)(pix1);
    X(free)(pix2);
    X(free)(mtch);
    X(cellfree)(dmres);
    X(free)(timing);
    X(free)(timing_tot);
    X(free)(timing_sock);
    X(free)(result);
    for(int igpu=0; igpu<ngpu; igpu++){
	hipSetDevice(gpus[igpu]);
	delete[] data[igpu]->event_w;
	delete[] data[igpu]->event_g;
	delete[] data[igpu]->event_p;
	hipDeviceReset();
    }
    free(data);
  
}
